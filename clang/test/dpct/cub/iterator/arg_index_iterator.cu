// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/iterator/constant_iterator %S/constant_iterator.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/iterator/constant_iterator/constant_iterator.dp.cpp %s

#include <hipcub/hipcub.hpp>
#include <iostream>

#define N 10

void init(int *&d_in, int *&d_out) {
  static constexpr int h_in[N] = {1, 2, 4, 8, 16, 32, 64, 128, 256, 512};
  hipMalloc((void **)&d_in, sizeof(int) * N);
  hipMalloc((void **)&d_out, sizeof(int) * N);
  hipMemcpy((void *)d_in, (void *)h_in, sizeof(h_in), hipMemcpyHostToDevice);
  std::cout << "Input:  ";
  std::copy(h_in, h_in + N, std::ostream_iterator<int>(std::cout, "\t"));
  std::cout << "\n";
}

void print(int *d_out) {
  int h_out[N];
  hipMemcpy((void *)h_out, (void *)d_out, sizeof(int) * N, hipMemcpyDeviceToHost);
  std::cout << "Output: ";
  std::copy(h_out, h_out + N, std::ostream_iterator<int>(std::cout, "\t"));
  std::cout << "\n";
}

using ArgIndexInputIterator = hipcub::ArgIndexInputIterator<int *>;
using Pair = ArgIndexInputIterator::value_type;

struct TransformOp {
  __device__ int operator()(const Pair &x) const {
    if (x.key & 1)
      return 0;
    return x.value;
  }
};

struct SumOp {
  __device__ int operator()(int x, int y) const {
    return x + y;
  }
};

void work() {
  int *d_in = nullptr;
  int *d_out = nullptr;
  int *d_temp = nullptr;
  size_t d_temp_size;
  init(d_in, d_out);
  SumOp scan_op;
  TransformOp input_iter_transform;
  auto input = hipcub::TransformInputIterator<int, TransformOp, ArgIndexInputIterator>(
    ArgIndexInputIterator(d_in), 
    input_iter_transform
  );
  hipcub::DeviceScan::InclusiveScan(d_temp, d_temp_size, input, d_out, scan_op, N);
  hipMalloc((void **)&d_temp, d_temp_size);
  hipcub::DeviceScan::InclusiveScan(d_temp, d_temp_size, input, d_out, scan_op, N);
  print(d_out);
}

int main() {
  work();
  return 0;
}

