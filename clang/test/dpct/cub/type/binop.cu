// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/type/binop %S/binop.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/type/binop/binop.dp.cpp --match-full-lines %s

#include <hipcub/hipcub.hpp>

int main() {
  
  return 0;
}
