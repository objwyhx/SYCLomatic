#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/blocklevel/blockreduce_p3 %S/blockreduce_p3.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/blocklevel/blockreduce_p3/blockreduce_p3.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/blocklevel/blockreduce_p3/blockreduce_p3.dp.cpp -o %T/blocklevel/blockreduce_p3/blockreduce_p3.dp.o %}

#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <iostream>

template <typename T>
__device__ __forceinline__ float reduce_topk_op_2(const float &a,
                                                  const float &b) {
  return a > b ? a : b;
}

__global__ void reduce_kernel(float *da) {
  typedef hipcub::BlockReduce<float, 32> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int id = threadIdx.x;
  BlockReduce rd(temp_storage);
  float temp = rd.Reduce(da[id], reduce_topk_op_2<float>);
  if (id == 0) {
    da[id] = temp;
  }
  __syncthreads();
}

__global__ void reduce_kernel1(float *da) {
  typedef hipcub::BlockReduce<float, 32> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int id = threadIdx.x;
  float temp = BlockReduce(temp_storage).Reduce(da[id], reduce_topk_op_2<float>);
  if (id == 0) {
    da[id] = temp;
  }
  __syncthreads();
}

template <class T, int THREAD_PRE_BLOCK>
__global__ void reduce_kernel_dependent(T *da) {
  typedef hipcub::BlockReduce<T, THREAD_PRE_BLOCK> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int id = threadIdx.x;
  BlockReduce rd(temp_storage);
  float temp = rd.Reduce(da[id], reduce_topk_op_2<float>);
  if (id == 0) {
    da[id] = temp;
  }
  __syncthreads();
}

template <class T, int THREAD_PRE_BLOCK>
__global__ void reduce_kernel_dependent1(T *da) {
  typedef hipcub::BlockReduce<T, THREAD_PRE_BLOCK> BlockReduce;
  __shared__ typename BlockReduce::TempStorage temp_storage;
  int id = threadIdx.x;
  float temp = BlockReduce(temp_storage).Reduce(da[id], reduce_topk_op_2<T>);
  if (id == 0) {
    da[id] = temp;
  }
  __syncthreads();
}

int main() {
  int N = 32;
  {
    float *ha = (float *)malloc(N * sizeof(float));
    float *da;
    hipMalloc(&da, N * sizeof(float));

    for (int i = 0; i < N; i++) {
      ha[i] = i * 1.0f;
    }

    hipMemcpy(da, ha, N * sizeof(float), hipMemcpyHostToDevice);
    reduce_kernel<<<1, 32>>>(da);
    reduce_kernel1<<<1, 32>>>(da);
    reduce_kernel_dependent<float, 32><<<1, 32>>>(da);
    reduce_kernel_dependent1<float, 32><<<1, 32>>>(da);
    hipMemcpy(ha, da, 1 * sizeof(float), hipMemcpyDeviceToHost);
    std::cout << ha[0] << std::endl;
    hipFree(da);
    free(ha);
  }

  {
    int *ha = (int *)malloc(N * sizeof(int));
    int *da;
    hipMalloc(&da, N * sizeof(int));

    for (int i = 0; i < N; i++) {
      ha[i] = i * 1.0f;
    }

    hipMemcpy(da, ha, N * sizeof(int), hipMemcpyHostToDevice);
    reduce_kernel_dependent<int, 32><<<1, 32>>>(da);
    reduce_kernel_dependent1<int, 32><<<1, 32>>>(da);
    hipMemcpy(ha, da, 1 * sizeof(int), hipMemcpyDeviceToHost);
    std::cout << ha[0] << std::endl;
    hipFree(da);
    free(ha);
  }


  return 0;
}
