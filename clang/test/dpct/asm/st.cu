// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/st %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/st/st.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/st/st.dp.cpp -o %T/st/st.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void st(int *a) {
  // CHECK: *((uint32_t *)(uintptr_t)a) = 111;
  asm volatile ("st.global.u32 [%0], %1;" :: "l"(a), "r"(111));
  // CHECK: *((uint32_t *)((uintptr_t)a + 4)) = 222;
  asm volatile ("st.global.u32 [%0 + 4], %1;" :: "l"(a), "r"(222));
  // CHECK: *((uint64_t *)((uintptr_t)a + 8)) = 0ull;
  asm volatile ("st.global.u64 [%0 + 8], %1;" :: "l"(a), "l"(0ull));
}

// clang-format on
