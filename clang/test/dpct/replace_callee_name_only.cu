// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.2, cuda-11.4
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.2, v11.4
// RUN: cat %s > %T/replace_callee_name_only.cu
// RUN: cat %S/replace_callee_name_only.yaml > %T/replace_callee_name_only.yaml
// RUN: cd %T
// RUN: rm -rf %T/replace_callee_name_only_output
// RUN: mkdir %T/replace_callee_name_only_output
// RUN: dpct -out-root %T/replace_callee_name_only_output replace_callee_name_only.cu --cuda-include-path="%cuda-path/include" --usm-level=none --rule-file=replace_callee_name_only.yaml -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/replace_callee_name_only_output/replace_callee_name_only.dp.cpp --match-full-lines replace_callee_name_only.cu

#include <hipcub/hipcub.hpp>
#include <stddef.h>

int n, *d_in, *d_out;
void *tmp;
size_t tmp_size;

#define CUB_WRAPPER(func, ...) do {                                       \
  void *temp_storage = nullptr;                                           \
  size_t temp_storage_bytes = 0;                                          \
  func(temp_storage, temp_storage_bytes, __VA_ARGS__);                    \
} while (false)

void test1() {
  // CHECK: CUB_WRAPPER(hipcub::DeviceScan::InclusiveSum, d_in, d_out, n);
  CUB_WRAPPER(hipcub::DeviceScan::InclusiveSum, d_in, d_out, n);
}

void test2() {
  // CHECK: hipcub::DeviceScan::InclusiveSum(tmp, tmp_size, d_in, d_out, n);
  hipcub::DeviceScan::InclusiveSum(tmp, tmp_size, d_in, d_out, n);
}
